#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "blur_interface.h"

// gaussian blur kernel in GPU constant memory
__constant__ float device_gaussian_blur_kernel[225];

#ifndef CUDA_CHECK
#define CUDA_CHECK(call) \
    do { \
        hipError_t _err = (call); \
        if (_err != hipSuccess) { \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ \
                      << ": " << hipGetErrorString(_err) << " (" << _err << ")\n"; \
            std::exit(1); \
        } \
    } while (0)
#endif

__global__ void gaussian_blur_kernel(const float* input, float* output, int width, int height, int kernel_dim) {
    // Get the current thread's position (pixel) in the grid.
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check: ensure we don't work on pixels outside the image.
    if (x >= width || y >= height) {
        return;
    }

    float sum = 0.0f;
    int radius = kernel_dim / 2;

    for (int row = 0; row < kernel_dim; row++) {
        for (int col = 0; col < kernel_dim; col++) {
            int neighbor_x = x + col - radius;
            int neighbor_y = y + row - radius;

            if (neighbor_x >= 0 && neighbor_x < width && neighbor_y >= 0 && neighbor_y < height) {
                int neighbor_idx = neighbor_y * width + neighbor_x;
                int kernel_idx = row * kernel_dim + col;
                sum += input[neighbor_idx] * device_gaussian_blur_kernel[kernel_idx];
            }
        }
    }

    int idx = y * width + x;
    output[idx] = sum;
}

void apply_gaussian_blur_cuda(const ImageData& host_img_input, ImageData& host_img_output, const float* host_gaussian_blur_kernel, int kernel_dim) {
    std::cout << "Applying Gaussian blur on CUDA" << std::endl;

    size_t data_size = (size_t)host_img_input.width * host_img_input.height * sizeof(float);

    float *device_img_input, *device_img_output;
    CUDA_CHECK(hipMalloc((void**)&device_img_input, data_size));
    CUDA_CHECK(hipMalloc((void**)&device_img_output, data_size));

    // pass the image data to the device
    CUDA_CHECK(hipMemcpy(device_img_input, host_img_input.data.get(), data_size, hipMemcpyHostToDevice));

    // pass the gaussian blur kernel matrix to the device
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(device_gaussian_blur_kernel), host_gaussian_blur_kernel, kernel_dim * kernel_dim * sizeof(float)));

    dim3 threads_per_block(16, 16);
    dim3 num_blocks((host_img_input.width + threads_per_block.x - 1) / threads_per_block.x,
                    (host_img_input.height + threads_per_block.y - 1) / threads_per_block.y);

    // execute the CUDA kernel in input host data
    gaussian_blur_kernel<<<num_blocks, threads_per_block>>>(device_img_input, device_img_output, host_img_input.width, host_img_input.height, kernel_dim);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // get the resulting computed output data from the device
    CUDA_CHECK(hipMemcpy(host_img_output.data.get(), device_img_output, data_size, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipFree(device_img_input));
    CUDA_CHECK(hipFree(device_img_output));

    std::cout << "Gaussian blur applied successfully" << std::endl;
}